#include "hip/hip_runtime.h"
#include "mymath.h"

#include <stdio.h>

__global__ void helloCUDA(float f) {
  printf("Hello world from thread %d with value %f\n", threadIdx.x, f);
}

void foosay(void) {
  printf("Foo-ing\n");
  helloCUDA<<<1, 5>>>(1.2345f);
  printf("Code: %d wanted %d\n", hipGetLastError(), 0);
  printf("Code: %d wanted %d\n", hipDeviceSynchronize(), 0);
  printf("Foo-ed\n");
  hipDeviceReset();
}
